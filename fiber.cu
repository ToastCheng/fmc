#include "header.h"

void fiber_initialization(Fibers* f)
{
	for(int i = 0; i < NUM_THREADS; i++)
	{
	    // initialization to zeros for data deposition 
		f[i].Exphoton[0] = 0;
		f[i].Fluoro[0] = 0;
		
		for(int j = 0; j <= NUM_OF_DETECTOR; j++)
		{
			f[i].up_data[j] = 0;
			f[i].down_data[j] = 0;

		}
		// normal configuration
		if(NORMAL)   
		{	
			// source fiber	
			f[i].radius[0]   = illumination_r;          		
			f[i].NA[0]       = NAOfSource;				
			f[i].angle[0]    = ANGLE*PI/180;
			f[i].position[0] = 0.0;			
			//first fiber,  SDS = 0.026 cm
			f[i].radius[1]   = collect_r;             
			f[i].NA[1]       = NAOfDetector;				
			f[i].position[1] = 0.022;                    
			f[i].angle[1]    = ANGLE*PI/180;
			//second fiber, SDS = 0.054 cm(before 20151029)
			f[i].radius[2]   = collect_r;          
			f[i].NA[1]       = NAOfDetector;				
			f[i].position[2] = 0.041;                    
			f[i].angle[2]    = ANGLE*PI/180;		
			//third fiber, SDS = 0.078 cm(before 20151103)
			f[i].radius[3]   = collect_r;             
			f[i].NA[3]       = NAOfDetector;				
			f[i].position[3] = 0.061;                    
			f[i].angle[3]    = ANGLE*PI/180;
			//fourth fiber, SDS = 0.049 cm
			f[i].radius[4]   = collect_r;             
			f[i].NA[4]       = NAOfDetector;				
			f[i].position[4] = 0.0215;                    
			f[i].angle[4]    = ANGLE*PI/180;
			//fourth fiber, SDS = 0.076 cm
			f[i].radius[5]   = collect_r;             
			f[i].NA[5]       = NAOfDetector;				
			f[i].position[5] = 0.045;                    
			f[i].angle[5]    = ANGLE*PI/180;
			//fourth fiber, SDS = 0.024 cm
			f[i].radius[6]   = collect_r;             
			f[i].NA[6]       = NAOfDetector;				
			f[i].position[6] = 0.073;                    
			f[i].angle[6]    = ANGLE*PI/180;
		}
		// oblique configuration
		else
		{
			for(int j = 1; j <= 3; j++)
			{
				f[i].radius[j]   = collect_r;	
				f[i].NA[j]       = NAOfDetector;		
				f[i].position[j] = 0.032*j;	
				f[i].angle[j]    = ANGLE*PI/180;
			}
		

			for(int j = 4; j <= 6; j++)
			{
				f[i].radius[j]   = collect_r;	
				f[i].NA[j]       = NAOfDetector;		
				f[i].position[j] = 0.022*(j-3);	
				f[i].angle[j]    = ANGLE*PI/180;
			}

			for(int j = 7; j <= 9; j++)
			{
				f[i].radius[j]   = collect_r;	
				f[i].NA[j]       = NAOfDetector;		
				f[i].position[j] = -0.022*(j-6);	
				f[i].angle[j]    = ANGLE*PI/180;
			}
		}
	}
}