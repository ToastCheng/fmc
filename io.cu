#include "hip/hip_runtime.h"
#include "header.h"
#include <cstdio>
#include <float.h>


void output_fiber(SimulationStruct* sim, float *up, float* down, float *Exp, float *F, int n)
{

	ofstream myfile, myfile_up, myfile_bt;

	char *sim_, *sim_up, *sim_bt;
	sim_ = (char*)malloc(30);
	sim_up = (char*)malloc(30);
	sim_bt = (char*)malloc(30);
	sim_[0] ='\0';
	sim_up[0] ='\0';
	sim_bt[0] ='\0';
	strcat(sim_,NUM_SPECTRUM);
	strcat(sim_,"/simulation");
	strcat(sim_,NUM_SPECTRUM);
	strcat(sim_,".txt");
	strcat(sim_up,NUM_SPECTRUM);
	strcat(sim_up,"/simulation_up");
	strcat(sim_up,NUM_SPECTRUM);
	strcat(sim_up,".txt");
	strcat(sim_bt,NUM_SPECTRUM);
	strcat(sim_bt,"/simulation_bt");
	strcat(sim_bt,NUM_SPECTRUM);
	strcat(sim_bt,".txt");
	myfile.open (sim_,ios::app);
	myfile_up.open(sim_up, ios::app);
	myfile_bt.open(sim_bt, ios::app);
	if(!myfile){
		 printf("failed to open %s\n",sim_);
		 exit(1);
	 }
	if(!myfile_up){
		 printf("failed to open %s\n",sim_);
		 exit(1);
	 }
	if(!myfile_bt){
		 printf("failed to open %s\n",sim_);
		 exit(1);
	 }

	double tmp;

	float scale1 = *Exp;    // #exitation photo

	// normal
	if(NORMAL == 1)
	{

		//fout2 << *Exp << " " << *F << endl;

		for(int i = 0; i < NUM_OF_DETECTOR; i++)
		{
			tmp = (double)(up[i]+down[i])/scale1;

			//fout << (double)(up[i]/scale1)/tmp << "\t" << (double)(down[i]/scale1)/tmp << "\t";
			//fout2 << (double)up[i]/scale1 << "\t" << (double)down[i]/scale1 << "\t";

			if (i==NUM_OF_DETECTOR-1){
				myfile << (double)(up[i] + down[i]) / scale1;
				myfile_up << (double)(up[i] / scale1);
				myfile_bt << (double)(down[i] / scale1);
			}
			else{
				myfile << (double)(up[i] + down[i]) / scale1 << "\t";
				myfile_up << (double)(up[i] / scale1) << "\t";
				myfile_bt << (double)(down[i] / scale1) << "\t";
			}
		}
	}
	// oblique
	else
	{
		for(int i = 0; i < 6; i++)
		{
			tmp = (double)(up[i]+down[i])/scale1;
			//fout << (double)(up[i]/scale1)/tmp << " " << (double)(down[i]/scale1)/tmp << " ";
			//fout2 << (double)up[i]/scale1 << " " << (double)down[i]/scale1 << " ";
			if (i == NUM_OF_DETECTOR - 1){ myfile << (double)(up[i] + down[i]) / scale1; }
			else{ myfile << (double)(up[i] + down[i]) / scale1 << "\t"; }
		}
	}

	//fout << endl;
	//fout2 << endl;
	myfile << endl;
	myfile_up << endl;
	myfile_bt << endl;

	myfile.close();
	myfile_up.close();
	myfile_bt.close();
	//fout.close();
	//fout2.close();
}

void read_anglepattern(float* gcumf, int nn)
{
	float gall[181] = {};
	for (int k = 0; k<181; k++)
		gcumf[k] = 0;
	float gprob[181] = {};

	// can be modified
	//char *address = "C:\\Users\\user\\Desktop\\GPU\\Anglepattern\\0.5\\";
	char filename[100];

	ifstream infile;
	switch (nn)
	{
		case 1:
			sprintf(filename, "input/tissue_anglepattern.txt"); //sprintf(filename,"%s0.5_%dnm.txt",address,index);  // can be modified
			//printf("Read tissue_anglepattern.txt\n");
			break;
		case 2:
			sprintf(filename, "input/tissue_anglepattern2.txt"); //sprintf(filename,"%s0.5_%dnm.txt",address,index);  // can be modified
			//printf("Read tissue_anglepattern2.txt\n");
			break;
	}
	infile.open(filename);
	if(!infile) printf("can't open anglepattern.txt!\n");
	float sum = 0;

	for (int i = 0; i < 181; i++)
	{
		infile >> gall[i];
		sum += gall[i] * sin(i*PI / 180);
	}

	for (int i = 0; i < 181; i++)
		gprob[i] = gall[i] * sin(i*PI / 180) / sum;

	for (int i = 0; i < 181; i++)
	{
		if (i == 0)
			gcumf[i] = gprob[i];
		else
			gcumf[i] += gcumf[i - 1] + gprob[i];
	}

	//printf("%e\n",gcumf[50]);

	infile.close();
}

//void read_anglepattern(int index, float* gcumf)
//{
//	float gall[181]={};
//	for(int k=0; k<181;k++)
//		gcumf[k] = 0;
//	float gprob[181]={};
//
//	// can be modified
//	//char *address = "C:\\Users\\user\\Desktop\\GPU\\Anglepattern\\0.5\\";
//	// complete address for the file
//	char filename[100];
//
//	ifstream infile;
//	sprintf(filename, "0.5_%dnm.txt", index); //sprintf(filename,"%s0.5_%dnm.txt",address,index);  // can be modified
//	infile.open(filename);
//
//	float sum = 0;
//
//	for(int i = 0; i < 181; i++)
//	{
//		infile >> gall[i];
//		sum += gall[i]*sin(i*PI/180);
//	}
//
//	for(int i = 0; i < 181; i++)
//		gprob[i] = gall[i]*sin(i*PI/180)/sum;
//
//	for(int i = 0; i < 181; i++)
//	{
//		if(i == 0)
//			gcumf[i] = gprob[i];
//		else
//			gcumf[i] += gcumf[i-1] + gprob[i];
//	}
//
//	//printf("%e\n",gcumf[50]);
//
//	infile.close();
//}

int read_data(SimulationStruct** simulations)
{
	// parameters to be modified
	unsigned long number_of_photons = NUMBER_PHOTONS;
	const int n_simulations = NUM_OF_SIMULATION;
	// double layer, default value = 2
	int n_layers = 2;
	// refractive index of outer medium	// water:1.33 tissue:1.58
	float medium_n = 1.60;
	// refractive index of tissue
	float tissue_n = 1.33;

	float start_weight;
	float upper_thickness;

	// read the file
	// anisotropy
	// directory address, maybe needs to be modified
	// can be modified
	//int excitation[1] = {365};
	// can be modified
	//int emmition[n_simulations] = {400,410,420,430,440,450,460,470,480,490,500,510,520,530,540,550,560,570,580,590,600,610,620,630,640,650};

	fstream myfile;
	char *para;
	para = (char*)malloc(30);
	para[0] = '\0';
	strcat(para,"DRSresult/");
	strcat(para,NUM_SPECTRUM);
	strcat(para,".txt");
	myfile.open (para,ios::in);
	if(!myfile) printf("failed to open %s!\n",para);
	float up_mua, up_mus, down_mua, down_mus;                      // excitation
	float up_mua_E[n_simulations],up_mus_E[n_simulations],         // emission
		  down_mua_E[n_simulations],down_mus_E[n_simulations];
	float up_quantum, down_quantum;                                // quantum yield
	float up_f_mua,   down_f_mua;                                  // mua of fluoro at excitation
	float up_emission_p[n_simulations], down_emission_p[n_simulations];
	float wavelength[n_simulations+1];



	myfile >> upper_thickness;     // input the thickness of upper layer
	myfile >> wavelength[0] >> up_mua >> up_mus >> down_mua >> down_mus;   // excitation parameters
	up_f_mua = up_mua;
	down_f_mua = down_mua;
	for(int i = 0; i < n_simulations; i++)   // emission parameters
	{
	    myfile >> wavelength[i+1] >> up_mua_E[i] >> up_mus_E[i]
		       >> down_mua_E[i] >> down_mus_E[i];
	}

	up_quantum = 1;
	down_quantum = 1;
	for(int i = 0; i < n_simulations; i++){   // emission probability at particular wavelength
	    up_emission_p[i] = 1;
		down_emission_p[i] = 1;
	}
	myfile.close();

	// Allocate memory for the SimulationStruct array
	*simulations = (SimulationStruct*) malloc(sizeof(SimulationStruct)*n_simulations);
	if(*simulations == NULL){perror("Failed to malloc simulations.\n");return 0;}

	for(int i = 0; i < n_simulations; i++)
	{
		(*simulations)[i].number_of_photons=number_of_photons;
		(*simulations)[i].n_layers = n_layers;

		// Allocate memory for the layers (including one for the upper and one for the lower)
		(*simulations)[i].layers = (LayerStruct*) malloc(sizeof(LayerStruct)*(n_layers+2));
		if((*simulations)[i].layers == NULL){perror("Failed to malloc layers.\n");return 0;}//{printf("Failed to malloc simulations.\n");return 0;}

		// Set upper refractive index (medium)
		(*simulations)[i].layers[0].n = medium_n;

		// Set the parameters of tissue (upper layer)
		(*simulations)[i].layers[1].n     = tissue_n;
		(*simulations)[i].layers[1].mua   = up_mua;
		(*simulations)[i].layers[1].muaE  = up_mua_E[i];
		// Set angle pattern
		read_anglepattern((*simulations)[i].layers[1].g, 1);   //read_anglepattern(excitation[0], (*simulations)[i].layers[1].g); //(*simulations)[i].layers[1].g     = read_anglepattern(excitation[0]);
		read_anglepattern((*simulations)[i].layers[1].gE, 1);  //read_anglepattern(emmition[i], (*simulations)[i].layers[1].gE); //(*simulations)[i].layers[1].gE    = read_anglepattern(emmition[i]);
		// Set other parameters
		(*simulations)[i].layers[1].z_min = 0;
		(*simulations)[i].layers[1].z_max = upper_thickness;
		(*simulations)[i].layers[1].mutr  = 1.0f/(up_mua+up_mus);
		(*simulations)[i].layers[1].mutrE = 1.0f/(up_mua_E[i]+up_mus_E[i]);
		(*simulations)[i].layers[1].f_mua = up_f_mua;
		(*simulations)[i].layers[1].quantum = up_quantum;
		(*simulations)[i].layers[1].emission_p = up_emission_p[i];

		// Set the parameters of tissue (lower layer)
		(*simulations)[i].layers[2].n     = tissue_n;
		(*simulations)[i].layers[2].mua   = down_mua;
		(*simulations)[i].layers[2].muaE  = down_mua_E[i];
		// Set angle pattern
		read_anglepattern((*simulations)[i].layers[2].g, 2);   //read_anglepattern(excitation[0], (*simulations)[i].layers[2].g);//(*simulations)[i].layers[2].g     = read_anglepattern(excitation[0]);
		read_anglepattern((*simulations)[i].layers[2].gE, 2);  //read_anglepattern(emmition[i], (*simulations)[i].layers[2].gE);//(*simulations)[i].layers[2].gE    = read_anglepattern(emmition[i]);
		// Set other parameters
		(*simulations)[i].layers[2].z_min = upper_thickness;
		//////////////////////////////////////////////////////////////////////////////
		//TOAST MODIFY 1 -> FLT_MAX
		(*simulations)[i].layers[2].z_max = FLT_MAX;				// set as infinity
		(*simulations)[i].layers[2].mutr  = 1.0f/(down_mua+down_mus);
		(*simulations)[i].layers[2].mutrE = 1.0f/(down_mua_E[i]+down_mus_E[i]);
		(*simulations)[i].layers[2].f_mua = down_f_mua;
		(*simulations)[i].layers[2].quantum = down_quantum;
		(*simulations)[i].layers[2].emission_p = down_emission_p[i];

		// Set lower refractive index (medium)
		(*simulations)[i].layers[n_layers+1].n = tissue_n;		// use "tissue_n" for no reflectance(assume that semi-infinity); use "medium_n" for layers

		//calculate start_weight
		float n1=n_source;
		float n2=(*simulations)[i].layers[1].n;
		float r = (n1-n2)/(n1+n2);
		r = r*r;
		start_weight = 1.0 * (1.0-r);
		//printf("Start weight=%e\n",start_weight);
		(*simulations)[i].start_weight=start_weight;

	}
	//system("pause");

	return n_simulations;
}
