#include "hip/hip_runtime.h"
#include "header.h"
#include <float.h> //for FLT_MAX


int InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim);
void FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem);
void FreeSimulationStruct(SimulationStruct* sim, int n_simulations);
__global__ void MCd(MemStruct DeviceMem, unsigned long long seed, int *n);
__global__ void LaunchPhoton_Global(MemStruct DeviceMem);
int InitDCMem(SimulationStruct* sim);
int Write_Simulation_Results(MemStruct* HostMem, SimulationStruct* sim, clock_t simulation_time);
int read_simulation_data(char* filename, SimulationStruct** simulations, int ignoreAdetection);
int interpret_arg(int argc, char* argv[], unsigned long long* seed, int* ignoreAdetection);

__device__ void LaunchPhoton(PhotonStruct* p, hiprandState *state);
__global__ void LaunchPhoton_Global(MemStruct DeviceMem, unsigned long long seed);
__device__ void fluoro_MC(PhotonStruct* p, hiprandState *state,Fibers *f);
__device__ void Spin(PhotonStruct*, float*,hiprandState* state);
__device__ unsigned int Reflect(PhotonStruct*, int, hiprandState* state);
__device__ unsigned int PhotonSurvive(PhotonStruct*, hiprandState* state);
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add);
__device__ void detect(PhotonStruct* p, Fibers* f);
__device__ int binarySearch(float *data, float value);
void fiber_initialization(Fibers* f);
void output_fiber(SimulationStruct* sim, float *up, float* down, float *Exp, float *F ,int n);
void calculate_reflectance(Fibers* f, float *up, float *down, float *Exp, float *F);

__device__ float rn_gen(hiprandState *s)
{
	float x = hiprand_uniform(s);
    return x;
}

void DoOneSimulation(SimulationStruct* simulation, int index)
{
	unsigned long long seed = time(NULL);
	float up_fluorescence[NUM_OF_DETECTOR] = {0};
	float down_fluorescence[NUM_OF_DETECTOR] = {0};
	float ExPhoton[1] = {0};
	float Fluoro[1] = {0};

	MemStruct DeviceMem;
	MemStruct HostMem;
	unsigned int threads_active_total=1;
	unsigned int i,ii;
	int  H_num_of_fluoro[1] = {0};
	int  *D_num_of_fluoro;
	hipMalloc(&D_num_of_fluoro, sizeof(int));
	hipMemcpy(D_num_of_fluoro, H_num_of_fluoro, sizeof(int),hipMemcpyHostToDevice);

    hipError_t cudastat;

	InitMemStructs(&HostMem,&DeviceMem,simulation);
	InitDCMem(simulation);

    dim3 dimBlock(NUM_THREADS_PER_BLOCK);
    dim3 dimGrid(NUM_BLOCKS);

	LaunchPhoton_Global<<<dimGrid,dimBlock>>>(DeviceMem, seed);
	hipDeviceSynchronize(); //CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
	cudastat=hipGetLastError(); // Check if there was an error
	if(cudastat)printf("Error code=%i, %s.\n",cudastat,hipGetErrorString(cudastat));

	i=0;

	while(threads_active_total>0)
	{
		i++;
		fiber_initialization(HostMem.f);
	    hipMemcpy(DeviceMem.f,HostMem.f,NUM_THREADS*sizeof(Fibers),hipMemcpyHostToDevice);

		//run the kernel
		seed = time(NULL);
		MCd<<<dimGrid,dimBlock>>>(DeviceMem, seed, D_num_of_fluoro);
		hipDeviceSynchronize(); //CUDA_SAFE_CALL( hipDeviceSynchronize() ); // Wait for all threads to finish
		cudastat=hipGetLastError(); // Check if there was an error
		if(cudastat)printf("Error code=%i, %s.\n",cudastat,hipGetErrorString(cudastat));

		// Copy thread_active from device to host, later deleted
		hipMemcpy(HostMem.thread_active, DeviceMem.thread_active, NUM_THREADS*sizeof(unsigned int), hipMemcpyDeviceToHost); //CUDA_SAFE_CALL(hipMemcpy(HostMem.thread_active,DeviceMem.thread_active,NUM_THREADS*sizeof(unsigned int),hipMemcpyDeviceToHost) );
		threads_active_total = 0;
		for(ii=0;ii<NUM_THREADS;ii++) threads_active_total+=HostMem.thread_active[ii];

		hipMemcpy(HostMem.num_terminated_photons, DeviceMem.num_terminated_photons, sizeof(unsigned int), hipMemcpyDeviceToHost); //CUDA_SAFE_CALL(hipMemcpy(HostMem.num_terminated_photons,DeviceMem.num_terminated_photons,sizeof(unsigned int),hipMemcpyDeviceToHost) );

		//printf("Run %u, Number of photons terminated %u, Threads active %u\n",i,*HostMem.num_terminated_photons,threads_active_total);

		hipMemcpy(HostMem.f, DeviceMem.f, NUM_THREADS*sizeof(Fibers), hipMemcpyDeviceToHost); //CUDA_SAFE_CALL(hipMemcpy(HostMem.f,DeviceMem.f,NUM_THREADS*sizeof(Fibers),hipMemcpyDeviceToHost));
		calculate_reflectance(HostMem.f,up_fluorescence, down_fluorescence, ExPhoton, Fluoro);
		hipMemcpy(&H_num_of_fluoro, D_num_of_fluoro, sizeof(int), hipMemcpyDeviceToHost);
	}
	//cout << "#" << index << " Simulation done!\n";
	//cout << *H_num_of_fluoro << "@@" << endl;

	output_fiber(simulation,up_fluorescence, down_fluorescence, ExPhoton, Fluoro, *H_num_of_fluoro);
	hipFree(D_num_of_fluoro);
	FreeMemStructs(&HostMem,&DeviceMem);
}

void calculate_reflectance(Fibers* f, float *up, float *down, float *ExP, float * F)
{
	for(int i = 0; i < NUM_THREADS; i++)
	{
		// normal configuration
		if(NORMAL)
		{
			ExP[0] += f[i].Exphoton[0];
			F[0] += f[i].Fluoro[0];

			for(int k = 1; k <= NUM_OF_DETECTOR; k++)
			{
				up[k-1] += f[i].up_data[k];
				down[k-1] += f[i].down_data[k];
			}

		}
		// oblique configuration
		else
		{
			ExP[0] += f[i].Exphoton[0];
			F[0] += f[i].Fluoro[0];

			up[0] += f[i].up_data[1];
			up[1] += f[i].up_data[2];
			up[2] += f[i].up_data[3];
			up[3] += f[i].up_data[4] + f[i].up_data[7];
			up[4] += f[i].up_data[5] + f[i].up_data[8];
			up[5] += f[i].up_data[6] + f[i].up_data[9];
			down[0] += f[i].down_data[1];
			down[1] += f[i].down_data[2];
			down[2] += f[i].down_data[3];
			down[3] += f[i].down_data[4] + f[i].down_data[7];
			down[4] += f[i].down_data[5] + f[i].down_data[8];
			down[5] += f[i].down_data[6] + f[i].down_data[9];
		}
	}
}

//Device function to add an unsigned integer to an unsigned long long using CUDA Compute Capability 1.1
__device__ void AtomicAddULL(unsigned long long* address, unsigned int add)
{
	if(atomicAdd((unsigned int*)address,add)+add<add)
		atomicAdd(((unsigned int*)address)+1,1u);
}

__global__ void MCd(MemStruct DeviceMem, unsigned long long seed, int* n)
{
    //Block index
    int bx = blockIdx.x;

    //Thread index
    int tx = threadIdx.x;

    //First element processed by the block
    int begin = NUM_THREADS_PER_BLOCK * bx;

	float s;	//step length

	float w_temp;

	PhotonStruct p = DeviceMem.p[begin+tx];
	//PhotonStruct fluoro = DeviceMem.fluoro[begin+tx];
	Fibers f = DeviceMem.f[begin+tx];

	int new_layer;

	hiprandState state = DeviceMem.state[begin+tx];
    hiprand_init(seed, begin+tx, 0, &state);

	//First, make sure the thread (photon) is active
	unsigned int ii = 0;
	if(!DeviceMem.thread_active[begin+tx]) ii = NUMSTEPS_GPU;

	for(;ii<NUMSTEPS_GPU;ii++) //this is the main while loop
	{
		if(layers_dc[p.layer].mutr!=FLT_MAX)
		{
			if (p.fluoro == false)
			s = -__logf(rn_gen(&state))*layers_dc[p.layer].mutr;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
			else
			s = -__logf(rn_gen(&state))*layers_dc[p.layer].mutrE;//sample step length [cm] //HERE AN OPEN_OPEN FUNCTION WOULD BE APPRECIATED
		}
		else
			s = 100.0f;//temporary, say the step in glass is 100 cm.

		//Check for layer transitions and in case, calculate s
		new_layer = p.layer;
		if(p.z+s*p.dz<layers_dc[p.layer].z_min){new_layer--; s = __fdividef(layers_dc[p.layer].z_min-p.z,p.dz);} //Check for upwards reflection/transmission & calculate new s
		if(p.z+s*p.dz>layers_dc[p.layer].z_max){new_layer++; s = __fdividef(layers_dc[p.layer].z_max-p.z,p.dz);} //Check for downward reflection/transmission

		p.x += p.dx*s;
		p.y += p.dy*s;
		p.z += p.dz*s;

		// 20150313
		if(p.z>layers_dc[p.layer].z_max)p.z=layers_dc[p.layer].z_max;
		if(p.z<layers_dc[p.layer].z_min)p.z=layers_dc[p.layer].z_min;

		if(new_layer!=p.layer)
		{
			// set the remaining step length to 0
			s = 0.0f;

			if(Reflect(&p,new_layer,&state)==0u)//Check for reflection
			{
				if (p.fluoro == false)
				{
					if(new_layer == 0)
					{ //Diffuse reflectance
						detect(&p,&f);
						p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
					}
					if(new_layer > *n_layers_dc)
					{	//Transmitted
						p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
					}
				}

				else
				{
					if(new_layer == 0)
					{ //Diffuse reflectance
						detect(&p,&f);
						p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
					}
					if(new_layer > *n_layers_dc)
					{	//Transmitted
						p.weight = 0; // Set the remaining weight to 0, effectively killing the photon
					}
				}
			}
		}

		if(s > 0.0f)
		{
			if (p.fluoro == false)
			{
				if(rn_gen(&state) < layers_dc[p.layer].mua*layers_dc[p.layer].mutr)  // absorption
				{
					// if absorbed by fluorophore
					// if(rn_gen(&state) < layers_dc[p.layer].f_mua/(layers_dc[p.layer].f_mua+layers_dc[p.layer].mua)) //20150303
					if(rn_gen(&state) < layers_dc[p.layer].f_mua/(layers_dc[p.layer].mua))
					{
						// if emitting fluorescence at particular wavelength
						if(rn_gen(&state) < layers_dc[p.layer].quantum * layers_dc[p.layer].emission_p)
						{
							*n = *n + 1;                             // count the number of fluorophore
							fluoro_MC(&p, &state, &f);
						}

						else
							p.weight = 0;
					}

					else
						p.weight = 0;
				}

				Spin(&p,layers_dc[p.layer].g,&state);
			}

			else
			{
				// weighted

				w_temp = layers_dc[p.layer].muaE*layers_dc[p.layer].mutrE*p.weight;
				p.weight -= w_temp;

				// fix weight
				/*
				if(rn_gen(&state) < layers_dc[p.layer].muaE*layers_dc[p.layer].mutrE)  // absorption
				{
					p.weight = 0;
				}
				*/
				Spin(&p,layers_dc[p.layer].gE,&state);
			}
		}

		if(!PhotonSurvive(&p,&state)) //if the photon doesn't survive
		{
			f.Exphoton[0] ++;

			if(atomicAdd(DeviceMem.num_terminated_photons,1u) < (*num_photons_dc-NUM_THREADS))
			{	// Ok to launch another photon
				LaunchPhoton(&p,&state);//Launch a new photon
			}
			else
			{	// No more photons should be launched.
				DeviceMem.thread_active[begin+tx] = 0u; // Set thread to inactive
				ii = NUMSTEPS_GPU;				// Exit main loop
			}
		}

	}//end main for loop!

	__syncthreads();	//necessary?

	//save the state of the MC simulation in global memory before exiting
	DeviceMem.p[begin+tx] = p;	//This one is incoherent!!!
	DeviceMem.f[begin+tx] = f;

}//end MCd

__device__ void fluoro_MC(PhotonStruct* p, hiprandState *state, Fibers *f)
{
	f->Fluoro[0] ++;

	float theta = 2 * PI * rn_gen(state);
    p->dz = -1 + 2 * rn_gen(state);
    p->dx = sqrt(1-p->dz*p->dz)*cos(theta);
    p->dy = sqrt(1-p->dz*p->dz)*sin(theta);

	p->fluoro = true;
	p->fluoro_layer = p->layer;
}

__device__ void LaunchPhoton(PhotonStruct* p, hiprandState *state)
{
	float rnd_Azimuth, rnd_direction, rnd_rotated;
	float AzimuthAngle;
	float launchPosition;
	float theta_direction;
	float rotated_angle;
	float uxprime, uyprime, uzprime;
	float angle = -ANGLE * PI / 180;

	rnd_Azimuth    = rn_gen(state);
	rnd_direction  = rn_gen(state);
	rnd_rotated    = rn_gen(state);
	AzimuthAngle   = 2 * PI * rnd_Azimuth;
	rotated_angle  = 2 * PI * rnd_rotated;

	float beam_width = illumination_r;  // 200 um, Gaussian beam profile

	launchPosition = beam_width*sqrt(-log(rn_gen(state))/2.0);

	p->x = launchPosition*cos(AzimuthAngle)/cos(angle);
	p->y = launchPosition*sin(AzimuthAngle);
	p->z = 0.0;

	theta_direction = asin(NAOfSource/n_source)*rnd_direction;
	p->dz = cos(theta_direction);
	p->dx = sin(theta_direction) * cos(rotated_angle);
	p->dy = sin(theta_direction) * sin(rotated_angle);

	uxprime = cos(angle)*p->dx - sin(angle)*p->dz;
	uyprime = sin(theta_direction)*sin(rotated_angle);
	uzprime = sin(angle)*p->dx + cos(angle)*p->dz;

	p->dx = uxprime, p->dy = uyprime, p->dz = uzprime;

	p->layer = 1;
	p->weight = *start_weight_dc; //specular reflection!
	p->fluoro = false;
	p->fluoro_layer = 0;
}

__global__ void LaunchPhoton_Global(MemStruct DeviceMem, unsigned long long seed)
{
	int bx=blockIdx.x;
    int tx=threadIdx.x;

    //First element processed by the block
    int begin=NUM_THREADS_PER_BLOCK*bx;

	PhotonStruct p;

	hiprandState state = DeviceMem.state[begin+tx];
    hiprand_init(seed, 0, 0, &state);

	LaunchPhoton(&p,&state);

	//__syncthreads();
	DeviceMem.p[begin+tx]=p;		//incoherent!?
}

/*
__device__ void Spin(PhotonStruct* p, float g, hiprandState *state)
{
	float theta, cost, sint;	// cosine and sine of the
						// polar deflection angle theta.
	float cosp, sinp;	// cosine and sine of the
						// azimuthal angle psi.
	float temp;
	float tempdir=p->dx;

	//This is more efficient for g!=0 but of course less efficient for g==0
	temp = __fdividef((1.0f-(g)*(g)),(1.0f-(g)+2.0f*(g)*rn_gen(state)));//Should be close close????!!!!!
	cost = __fdividef((1.0f+(g)*(g) - temp*temp),(2.0f*(g)));
	if(g==0.0f)
		cost = 2.0f*rn_gen(state)-1.0f;//Should be close close??!!!!!

	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rn_gen(state),&sinp,&cosp);// spin psi [0-2*PI)

	temp = sqrtf(1.0f - p->dz*p->dz);

	if(temp==0.0f) //normal incident.
	{
		p->dx = sint*cosp;
		p->dy = sint*sinp;
		p->dz = copysignf(cost,p->dz*cost);
	}
	else // regular incident.
	{
		p->dx = __fdividef(sint*(p->dx*p->dz*cosp - p->dy*sinp),temp) + p->dx*cost;
		p->dy = __fdividef(sint*(p->dy*p->dz*cosp + tempdir*sinp),temp) + p->dy*cost;
		p->dz = -sint*cosp*temp + p->dz*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp=rsqrtf(p->dx*p->dx+p->dy*p->dy+p->dz*p->dz);
	p->dx = p->dx*temp;
	p->dy = p->dy*temp;
	p->dz = p->dz*temp;
}// end Spin
*/
__device__ int binarySearch(float *data, float value)
{
    int middle;
	int left = 0, right = 180;
    while (left <= right)
    {
        middle = (right + left) / 2;

        if (data[middle] == value)
            return middle;

        if (data[middle] > value)
            right = middle - 1;
        else
            left = middle + 1;
    }
	if (data[middle] > value)
	    return middle;
	else
		return middle + 1;
}

__device__ void Spin(PhotonStruct* p, float *g, hiprandState *state)
{
	float theta, cost, sint;	// cosine and sine of the
						// polar deflection angle theta.
	float cosp, sinp;	// cosine and sine of the
						// azimuthal angle psi.
	float temp;
	float tempdir=p->dx;

	float rn = rn_gen(state);
    int sample;

	sample = binarySearch(g,rn);

	theta = sample-1+__fdividef((rn-g[sample-1]),(g[sample]-g[sample-1]));
    theta = __fdividef(theta*PI,180);
	cost = cos(theta);

	sint = sqrtf(1.0f - cost*cost);

	__sincosf(2.0f*PI*rn_gen(state),&sinp,&cosp);// spin psi [0-2*PI)

	temp = sqrtf(1.0f - p->dz*p->dz);

	if(temp==0.0f) //normal incident.
	{
		p->dx = sint*cosp;
		p->dy = sint*sinp;
		p->dz = copysignf(cost,p->dz*cost);
	}
	else // regular incident.
	{
		p->dx = __fdividef(sint*(p->dx*p->dz*cosp - p->dy*sinp),temp) + p->dx*cost;
		p->dy = __fdividef(sint*(p->dy*p->dz*cosp + tempdir*sinp),temp) + p->dy*cost;
		p->dz = -sint*cosp*temp + p->dz*cost;
	}

	//normalisation seems to be required as we are using floats! Otherwise the small numerical error will accumulate
	temp=rsqrtf(p->dx*p->dx+p->dy*p->dy+p->dz*p->dz);
	p->dx = p->dx*temp;
	p->dy = p->dy*temp;
	p->dz = p->dz*temp;
}// end Spin



__device__ unsigned int Reflect(PhotonStruct* p, int new_layer, hiprandState *state)
{
	//Calculates whether the photon is reflected (returns 1) or not (returns 0)
	// Reflect() will also update the current photon layer (after transmission) and photon direction (both transmission and reflection)

	float n1 = layers_dc[p->layer].n;
	float n2 = layers_dc[new_layer].n;
	float r;
	float cos_angle_i = fabsf(p->dz);

	if(n1==n2)//refraction index matching automatic transmission and no direction change
	{
		p->layer = new_layer;
		return 0u;
	}

	if(n1>n2 && n2*n2<n1*n1*(1-cos_angle_i*cos_angle_i))//total internal reflection, no layer change but z-direction mirroring
	{
		p->dz *= -1.0f;
		return 1u;
	}

	if(cos_angle_i==1.0f)//normal incident
	{
		r = __fdividef((n1-n2),(n1+n2));
		if(rn_gen(state)<=r*r)
		{
			//reflection, no layer change but z-direction mirroring
			p->dz *= -1.0f;
			return 1u;
		}
		else
		{	//transmission, no direction change but layer change
			p->layer = new_layer;
			return 0u;
		}
	}

	//gives almost exactly the same results as the old MCML way of doing the calculation but does it slightly faster
	// save a few multiplications, calculate cos_angle_i^2;
	float e = __fdividef(n1*n1,n2*n2)*(1.0f-cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
	r=2*sqrtf((1.0f-cos_angle_i*cos_angle_i)*(1.0f-e)*e*cos_angle_i*cos_angle_i);//use r as a temporary variable
	e=e+(cos_angle_i*cos_angle_i)*(1.0f-2.0f*e);//Update the value of e
	r = e*__fdividef((1.0f-e-r),((1.0f-e+r)*(e+r)));//Calculate r

	if(rn_gen(state)<=r)
	{
		// Reflection, mirror z-direction!
		p->dz *= -1.0f;
		return 1u;
	}
	else
	{
		// Transmission, update layer and direction
		r = __fdividef(n1,n2);
		e = r*r*(1.0f-cos_angle_i*cos_angle_i); //e is the sin square of the transmission angle
		p->dx *= r;
		p->dy *= r;
		p->dz = copysignf(sqrtf(1-e) ,p->dz);
		p->layer = new_layer;
		return 0u;
	}

}

__device__ unsigned int PhotonSurvive(PhotonStruct* p, hiprandState *state)
{
	//Calculate wether the photon survives (returns 1) or dies (returns 0)

	if(p->weight>WEIGHTI) return 1u; // No roulette needed
	if(p->weight==0) return 0u;	// Photon has exited slab, i.e. kill the photon

	if(rn_gen(state) < CHANCE)
	{
		//p->weight = __float2uint_rn(__fdividef((float)p->weight,CHANCE));
		p->weight = __fdividef((float)p->weight,CHANCE);
		return 1u;
	}
	return 0u;
}

__device__ void detect(PhotonStruct* p, Fibers* f)
{
	float angle = ANGLE*PI/180;
	float critical = asin(f->NA[1]/ n_detector);
    float uz_rotated=(p->dx*sin(angle))+(p->dz*cos(angle));
	float uz_angle = acos(fabs(uz_rotated));
	float distance;

	if(uz_angle <= critical)  // successfully detected
	{
		if(NORMAL)   // normal configuration
		{
			if(p->fluoro == true)
			{
				// ISS circle
				/*
				for(int i = 1; i <= 3 ; i++)
				{
					if(pow((p->x-f->position[i])*cos(angle),2) + pow(p->y,2) <= f->radius[i]*f->radius[i])
					{
						if(p->fluoro_layer==1)
							f->up_data[i] += p->weight;
						else
							f->down_data[i] += p->weight;
					}
				}

				for(int i = 4; i <= 6 ; i++)
				{
					if(pow((p->y-f->position[i]),2) + pow(p->x*cos(angle),2) <= f->radius[i]*f->radius[i])
					{
						if(p->fluoro_layer==1)
							f->up_data[i] += p->weight;
						else
							f->down_data[i] += p->weight;
					}

				}
				*/

				// ISS annular
				distance = sqrt(p->x * p->x + p->y * p->y);

				for(int i = 1; i <= 6 ; i++)
				{
					if((distance>=(f->position[i]-f->radius[i])) && (distance<=(f->position[i]+f->radius[i])))
					{
						float temp;
						temp = (distance*distance + f->position[i]*f->position[i] - f->radius[i]*f->radius[i])/(2*distance*f->position[i]);
						// check for rounding error!
						if(temp > 1.0f)
							temp = 1.0f;

						if(p->fluoro_layer==1)
							f->up_data[i] += p->weight * acos(temp) * RPI;
						else
							f->down_data[i] += p->weight * acos(temp) * RPI;

					}
				}

			}

		}
		// oblique configuration
		else
		{

		}
	}
    return;
}

int InitDCMem(SimulationStruct* sim)
{
	// Copy num_photons_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(n_layers_dc),&(sim->n_layers),sizeof(unsigned int));

	// Copy start_weight_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(start_weight_dc),&(sim->start_weight),sizeof(float));

	// Copy layer data to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(layers_dc),sim->layers,(sim->n_layers+2)*sizeof(LayerStruct));

	// Copy num_photons_dc to constant device memory
	hipMemcpyToSymbol(HIP_SYMBOL(num_photons_dc),&(sim->number_of_photons),sizeof(unsigned int));

	return 0;
}

int InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{
	// Allocate p on the device!!
	hipMalloc((void**)&DeviceMem->p,NUM_THREADS*sizeof(PhotonStruct));

	// Allocate fluoro on the device!!
	// hipMalloc((void**)&DeviceMem->fluoro,NUM_THREADS*sizeof(PhotonStruct));

	// Allocate thread_active on the device and host
	HostMem->thread_active = (unsigned int*) malloc(NUM_THREADS*sizeof(unsigned int));
	if(HostMem->thread_active==NULL){printf("Error allocating HostMem->thread_active"); exit (1);}
	for(int i=0;i<NUM_THREADS;i++)HostMem->thread_active[i]=1u;

	hipMalloc((void**)&DeviceMem->thread_active,NUM_THREADS*sizeof(unsigned int));
	hipMemcpy(DeviceMem->thread_active,HostMem->thread_active,NUM_THREADS*sizeof(unsigned int),hipMemcpyHostToDevice);

	//Allocate num_launched_photons on the device and host
	HostMem->num_terminated_photons = (unsigned int*) malloc(sizeof(unsigned int));
	if(HostMem->num_terminated_photons==NULL){printf("Error allocating HostMem->num_terminated_photons"); exit (1);}
	*HostMem->num_terminated_photons=0;

	hipMalloc((void**)&DeviceMem->num_terminated_photons,sizeof(unsigned int));
	hipMemcpy(DeviceMem->num_terminated_photons,HostMem->num_terminated_photons,sizeof(unsigned int),hipMemcpyHostToDevice);

	//Allocate and initialize fiber f on the device and host
	HostMem->f = (Fibers*) malloc(NUM_THREADS*sizeof(Fibers));
	hipMalloc((void**)&DeviceMem->f,NUM_THREADS*sizeof(Fibers));
	fiber_initialization(HostMem->f);
	hipMemcpy(DeviceMem->f,HostMem->f,NUM_THREADS*sizeof(Fibers),hipMemcpyHostToDevice);

	//Allocate states on the device and host
	hipMalloc((void**)&DeviceMem->state,NUM_THREADS*sizeof(hiprandState));


	return 1;
}

void FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem)
{
	free(HostMem->thread_active);
	free(HostMem->num_terminated_photons);
	free(HostMem->f);

	hipFree(DeviceMem->thread_active);
	hipFree(DeviceMem->num_terminated_photons);
	hipFree(DeviceMem->f);
	hipFree(DeviceMem->state);
}
